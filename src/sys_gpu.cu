#include "../lib/sys.hpp"
#include <iostream>
#include <hip/hip_runtime_api.h>

int System::_gpu_arch(int major, int minor) {
    // defs for GPU arch types using streaming multiprocessor to determine
    // # of cores per proc
    struct ARCHInfo {
        int sm;  // 0xMm (hex), M = SM Major version, and m = SM minor version
        int cores;
    };

    ARCHInfo cores_per_proc[] = {
        {0x10, 8},  // Tesla Generation (SM 1.0) G80 class
        {0x11, 8},  // Tesla Generation (SM 1.1) G8x class
        {0x12, 8},  // Tesla Generation (SM 1.2) G9x class
        {0x13, 8},  // Tesla Generation (SM 1.3) GT200 class
        {0x20, 32}, // Fermi Generation (SM 2.0) GF100 class
        {0x21, 48}, // Fermi Generation (SM 2.1) GF10x class
        {0x30, 192},// Kepler Generation (SM 3.0) GK10x class
        {0x32, 192},// Kepler Generation (SM 3.2) GK20A class
        {0x35, 192},// Kepler Generation (SM 3.5) GK11x class
        {0x37, 192},// Kepler Generation (SM 3.7) GK21x class
        {0x50, 128},// Maxwell Generation (SM 5.0) GM10x class
        {0x52, 128},// Maxwell Generation (SM 5.2) GM20x class
        {0x53, 128},// Maxwell Generation (SM 5.3) GM20x class
        {0x60, 64}, // Pascal Generation (SM 6.0) GP100 class
        {0x61, 128},// Pascal Generation (SM 6.1) GP10x class
        {0x62, 128},// Pascal Generation (SM 6.2) GP10x class
        {0x70, 64}, // Volta Generation (SM 7.0) GV100 class
        {0x72, 64}, // Volta Generation (SM 7.2) GV10B class
        {0x75, 64}, // Turing Generation (SM 7.5) TU10x class
        {0x80, 64}, // Ampere Generation (SM 8.0) GA100 class
        {0x86, 64}, // Ampere Generation (SM 8.6) GA10x class
        // add more archs here???
        {-1, -1}    // Indicates unknown SM
    };

    int index = 0;
    while (cores_per_proc[index].sm != -1) {
        if (cores_per_proc[index].sm == ((major << 4) + minor)) {
            return cores_per_proc[index].cores;
        }
        index++;
    }

    // if we get here we got an invalid arch
    return -1;
}

void System::gpu_info() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No CUDA-compatible GPU found." << std::endl;
        return;
    }

    // Assuming you want information about the first GPU
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    int nvd_driver_version, cuda_version;
    size_t gpu_mem_total, gpu_mem_free;

    /** GPU DEVICE INFORMATION */
    System::name                = deviceProp.name;

    hipDriverGetVersion(&nvd_driver_version);
    hipRuntimeGetVersion(&cuda_version);
    System::nvd_driver_version  = nvd_driver_version;
    System::cuda_version        = cuda_version;
    System::major               = deviceProp.major;
    System::minor               = deviceProp.minor;
    
    System::shared_mem_pb       = deviceProp.sharedMemPerBlock;
    System::total_const_mem     = deviceProp.totalConstMem;
    System::total_glbl_mem      = static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f);
    hipMemGetInfo(&gpu_mem_free, &gpu_mem_total);
    
    System::gpu_mem_total       = static_cast<float>(gpu_mem_total / 1048576.0f);
    System::gpu_mem_free        = static_cast<float>(gpu_mem_free / 1048576.0f);
    System::gpu_mem_used        = static_cast<float>(gpu_mem_total - gpu_mem_free);

    System::gpu_mp_count        = deviceProp.multiProcessorCount;
    System::cuda_cores          = _gpu_arch(deviceProp.major, 
                                                        deviceProp.minor);
    System::max_clock_rt        = deviceProp.clockRate * 1e-6f;
    
    System::max_mp_threads      = deviceProp.maxThreadsPerMultiProcessor;
    System::max_pb_threads      = deviceProp.maxThreadsPerBlock;
    
    System::max_dim_threads[0]  = deviceProp.maxThreadsDim[0];
    System::max_dim_threads[1]  = deviceProp.maxThreadsDim[1];
    System::max_dim_threads[2]  = deviceProp.maxThreadsDim[2];
    System::max_grid_size[0]    = deviceProp.maxGridSize[0];
    System::max_grid_size[1]    = deviceProp.maxGridSize[1];
    System::max_grid_size[2]    = deviceProp.maxGridSize[2];
}

void System::gpu_info_print() {
    std::cout << "\nDevice: \"" << System::name << "\"" << std::endl;

    std::cout << "  CUDA Driver Version / Runtime Version          "
              << System::nvd_driver_version / 1000 << "."
              << (System::nvd_driver_version % 100) / 10
              << " / " << System::cuda_version / 1000 << "."
              << (System::cuda_version % 100) / 10 << std::endl;

    std::cout << "  CUDA Capability Major/Minor version number:    "
              << System::major << "." << System::minor << std::endl;

    std::cout << "  Total amount of global memory:                 "
              << System::total_glbl_mem << " MBytes" << std::endl;

    std::cout << "  ( " << System::gpu_mp_count << ") Multiprocessors, ("
              << System::gpu_mp_count * System::cuda_cores
              << ") CUDA Cores/MP:     " << System::cuda_cores
              << " CUDA Cores" << std::endl;

    std::cout << "  GPU Max Clock rate:                            "
              << System::max_clock_rt << " GHz" << std::endl;

    std::cout << "  Total amount of constant memory:               "
              << System::total_const_mem << " bytes" << std::endl;

    std::cout << "  Total amount of shared memory per block:       "
              << System::shared_mem_pb << " bytes" << std::endl;

    std::cout << "  Maximum number of threads per multiprocessor:  "
              << System::max_mp_threads << std::endl;

    std::cout << "  Maximum number of threads per block:           "
              << System::max_pb_threads << std::endl;

    std::cout << "  Max dimension size of a thread block (x,y,z): ("
              << System::max_dim_threads[0] << ", " << System::max_dim_threads[1] << ", "
              << System::max_dim_threads[2] << ")" << std::endl;

    std::cout << "  Max dimension size of a grid size    (x,y,z): ("
              << System::max_grid_size[0] << ", " << System::max_grid_size[1] << ", "
              << System::max_grid_size[2] << ")" << std::endl;

    std::cout << "  GPU MEMORY TOTAL" << System::gpu_mem_total << "\n";

    std::cout << "  GPU MEMORY USED" << System::gpu_mem_used << "\n";

    std::cout << "  GPU MEMORY FREE" << System::gpu_mem_free << "\n";

}

